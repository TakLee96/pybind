#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>

#include "kernel.h"

namespace mumpy::cuda {

#define CHECK(expr)                            \
  do {                                         \
    if (!(expr)) {                             \
      std::cerr << "CHECK failed: " << #expr;  \
      exit(1);                                 \
    }                                          \
  } while (0)

#define CUDA_CHECK(expr)                                                \
  do {                                                                  \
    hipError_t err = (expr);                                           \
    if (err != hipSuccess) {                                           \
      std::cerr << "CUDA_CHECK failed: " << #expr                       \
                << "\nCUDA Error Code" << err                           \
                << "\nError String:" << hipGetErrorString(err);        \
      exit(err);                                                        \
    }                                                                   \
  } while (0)

__global__ void vector_add_kernel(const float *A, const float *B, float *C,
                                  int num_elements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < num_elements) {
    C[i] = A[i] + B[i];
  }
}

Eigen::VectorXf vector_add(const Eigen::VectorXf& x, const Eigen::VectorXf& y) {
  CHECK(x.size() == y.size());
  Eigen::VectorXf z;
  z.resizeLike(x);
  z.setZero();

  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;
  int num_elements = x.size();
  size_t size = num_elements * sizeof(float);

  // Allocate the device inputs and outputs
  float *d_A = NULL;
  CUDA_CHECK(hipMalloc((void **)&d_A, size));
  float *d_B = NULL;
  CUDA_CHECK(hipMalloc((void **)&d_B, size));
  float *d_C = NULL;
  CUDA_CHECK(hipMalloc((void **)&d_C, size));

  // Copy and launch
  CUDA_CHECK(hipMemcpy(d_A, x.data(), size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B, y.data(), size, hipMemcpyHostToDevice));
  int threadsPerBlock = 256;
  int blocksPerGrid = (num_elements + threadsPerBlock - 1) / threadsPerBlock;
  vector_add_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, num_elements);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipMemcpy(z.data(), d_C, size, hipMemcpyDeviceToHost));

  // Free device global memory
  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));
  CUDA_CHECK(hipFree(d_C));

  return z;
}

} // mumpy::cuda
