#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>

#include "kernel.h"

namespace mumpy::cuda {

#define CHECK(expr)                            \
  do {                                         \
    if (!(expr)) {                             \
      std::cerr << "CHECK failed: " << #expr;  \
      exit(1);                                 \
    }                                          \
  } while (0)

#define CUDA_CHECK(expr)                                                \
  do {                                                                  \
    hipError_t err = (expr);                                           \
    if (err != hipSuccess) {                                           \
      std::cerr << "CUDA_CHECK failed: " << #expr                       \
                << "\nCUDA Error Code" << err                           \
                << "\nError String:" << hipGetErrorString(err);        \
      exit(err);                                                        \
    }                                                                   \
  } while (0)

__global__ void vector_add_kernel(const double *A, const double *B, double *C,
                                  int num_elements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < num_elements) {
    C[i] = A[i] + B[i];
  }
}

Eigen::VectorXd vector_add(const Eigen::VectorXd& x, const Eigen::VectorXd& y) {
  CHECK(x.size() == y.size());
  Eigen::VectorXd z;
  z.resizeLike(x);
  z.setZero();

  // Allocate the device inputs and outputs
  int num_elements = x.size();
  size_t size = num_elements * sizeof(double);
  double *d_A = nullptr;
  CUDA_CHECK(hipMalloc((void **)&d_A, size));
  double *d_B = nullptr;
  CUDA_CHECK(hipMalloc((void **)&d_B, size));
  double *d_C = nullptr;
  CUDA_CHECK(hipMalloc((void **)&d_C, size));

  // Copy and launch
  CUDA_CHECK(hipMemcpy(d_A, x.data(), size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B, y.data(), size, hipMemcpyHostToDevice));
  int threadsPerBlock = 256;
  int blocksPerGrid = (num_elements + threadsPerBlock - 1) / threadsPerBlock;
  vector_add_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, num_elements);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipMemcpy(z.data(), d_C, size, hipMemcpyDeviceToHost));

  // Free device global memory
  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));
  CUDA_CHECK(hipFree(d_C));

  return z;
}

__global__ void matmul_kernel(const double *A, const double *B, double *C, int m, int n, int k) { 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    double sum = 0;
    if (col < k && row < m) {
      for (int i = 0; i < n; i++) {
        sum += A[row * n + i] * B[i * k + col];
      }
      C[row * k + col] = sum;
    }
}

MatrixXdRowMajor matmul(const MatrixXdRowMajor& x, const MatrixXdRowMajor& y) {
  CHECK(x.cols() == y.rows());
  int m = x.rows();
  int n = x.cols();
  int k = y.cols();
  MatrixXdRowMajor z(m, k);
  z.setZero();

  // Allocate the device inputs and outputs
  double *d_A = nullptr;
  CUDA_CHECK(hipMalloc((void **)&d_A, x.size() * sizeof(double)));
  double *d_B = nullptr;
  CUDA_CHECK(hipMalloc((void **)&d_B, y.size() * sizeof(double)));
  double *d_C = nullptr;
  CUDA_CHECK(hipMalloc((void **)&d_C, z.size() * sizeof(double)));

  // Copy and launch
  CUDA_CHECK(hipMemcpy(d_A, x.data(), x.size() * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B, y.data(), y.size() * sizeof(double), hipMemcpyHostToDevice));
  int block_size = 16;
  unsigned int grid_rows = (m + block_size - 1) / block_size;
  unsigned int grid_cols = (k + block_size - 1) / block_size;
  dim3 dimGrid(grid_cols, grid_rows);
  dim3 dimBlock(block_size, block_size);
  matmul_kernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, m, n, k);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipMemcpy(z.data(), d_C, z.size() * sizeof(double), hipMemcpyDeviceToHost));

  // Free device global memory
  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));
  CUDA_CHECK(hipFree(d_C));

  return z;
}

} // mumpy::cuda
